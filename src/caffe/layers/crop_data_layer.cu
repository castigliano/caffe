#include <vector>
#include "caffe/data_layers.hpp"

namespace caffe {

template <typename Dtype>
void CropDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
	//LOG(INFO) << "I AM AT WORK ####################### GPU ###################";
	Batch<Dtype>* batch = this->prefetch_full_.pop("Data layer prefetch queue empty");
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
             top[0]->mutable_gpu_data());
  DLOG(INFO) << "Prefetch copied";
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
	//New thread will be created by GenericWindowData Layer
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  this->prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(CropDataLayer);

}  // namespace caffe
